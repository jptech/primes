
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define THREADS_PER_BLOCK (1024)
#define NUM_BLOCKS (64)

void usage()
{
	printf("Usage: prime_cpu N\n");
	exit(0);
}

__global__ void sum_primes( int *N, unsigned long long *sum )
{
	unsigned int i, j, notprime;
	unsigned int maxj = 0;
	unsigned int maxN = *N;
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long long block_sum = 0;

	__shared__ unsigned long long sums[THREADS_PER_BLOCK];

	sums[threadIdx.x] = 0;

	i = index;
	if(i < 2) i += (blockDim.x * NUM_BLOCKS);

	for(; i < maxN; i += (blockDim.x * NUM_BLOCKS))
	{
		notprime = 0;
		maxj = sqrt( (double) i);
		for(j = 2; j <= maxj; j++)
		{
			if(i % j == 0)
			{
				notprime = 1;
				break;
			}
		}
		if(notprime == 0) sums[threadIdx.x] += i;
	}

	__syncthreads();

	if(threadIdx.x == 0)
	{
		for(i = 0; i < THREADS_PER_BLOCK; i++)
		{
			block_sum += sums[i];
		}
		atomicAdd(sum, block_sum);
	}

}

int main(int argc, char **argv)
{
	int N;
	int num_threads, num_blocks;
	int *n_cuda;
	unsigned long long *sum_cuda;
	unsigned long long sum;
	struct timeval tv;
	double t0, t1;

	if(argc != 2)
		usage();

	N = atoi(argv[1]);

	gettimeofday(&tv, NULL);
	t0 = tv.tv_usec;
	t0 /= 1000000.0;
	t0 += tv.tv_sec;

	num_blocks = NUM_BLOCKS;
	num_threads = THREADS_PER_BLOCK;
	sum = 0;

	hipMalloc( (void **)&n_cuda, sizeof(int) );
	hipMalloc( (void **)&sum_cuda, sizeof(unsigned long long) );

	hipMemcpy( n_cuda, (void *) &N, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( sum_cuda, (void *) &sum, sizeof(unsigned long long), hipMemcpyHostToDevice );

	sum_primes <<< num_blocks, num_threads >>> ( n_cuda, sum_cuda );

	hipMemcpy( &sum, sum_cuda, sizeof(unsigned long long), hipMemcpyDeviceToHost );

	gettimeofday(&tv, NULL);
	t1 = tv.tv_usec;
	t1 /= 1000000.0;
	t1 += tv.tv_sec;

	printf("N: %d\n", N);
	printf("sum of primes up to N: %ld\n", sum);
	printf("Time elapsed: %lf\n", t1 - t0);

	hipFree(n_cuda);
	hipFree(sum_cuda);

	return 0;
}
